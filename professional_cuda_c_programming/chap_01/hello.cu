/**
 * @brief	A simple hello world program utilizing GPU capabilities
 *
 * @build	nvcc -gencode arch=compute_87,code=sm_87 hello.cu -o hello
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * @brief	Function to get executed on the GPU
 */
__global__ void helloFromGPU (void)
{
	printf("Hello World from the GPU!\n");
}

/**
 * @brief	Driver function. To run on CPU
 */
int main(void)
{
	// hello from CPU
	printf("Hello World from CPU!\n");

	helloFromGPU <<<1, 10>>>();
	hipDeviceReset();
	return 0;
}
