#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * @brief		Driver function
 */
int main(int argc, char **argv)
{
	// Define total data elements
	int nElem = 1024;

	// Define grid and block structure
	dim3 block(1024);
	dim3 grid((nElem + block.x - 1) / block.x);
	printf("grid.x: %d, block.x: %d\n", grid.x, block.x);

	// Reset block
	block.x = 512;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x: %d, block.x: %d\n", grid.x, block.x);

	// Reset block
	block.x = 256;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x: %d, block.x: %d\n", grid.x, block.x);

	// Reset block
	block.x = 128;
	grid.x = (nElem + block.x - 1) / block.x;
	printf("grid.x: %d, block.x: %d\n", grid.x, block.x);

	// Finally, reset the device before you leave
	hipDeviceReset();
	
	return 0;
}
